
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <random>

#define K 2

#define THREAD_PER_BLOCK 512
#define A 1000 //alias for N set at compile time...

int N = 0;

std::vector<std::string> split (const std::string &s, char delim) {
    std::vector<std::string> result;
    std::stringstream ss (s);
    std::string item;

    while (getline (ss, item, delim)) {
        result.push_back (item);
    }

    return result;
}



float* loadCsv(const std::string& fileName){
    std::string line;
    std::ifstream file(fileName);
    std::string word;
    if(!file.is_open()){
        std::cout << "error opening file." << std::endl;
        return nullptr;
    }
    int lineNumber = 0;
    while(getline(file,line)){
        lineNumber += 1;
    }
    N = lineNumber;
    file.close();

    std::ifstream file1(fileName);
    float *dataPoints = (float*) malloc(sizeof(float)*lineNumber*4);
    int i = 0;
    while(getline(file1,line)){
        std::vector<std::string> coords = split(line,',');
        dataPoints[i++] = stof(coords[0]);
		dataPoints[i++] = stof(coords[1]);
        dataPoints[i++] = stof(coords[2]);
		dataPoints[i++] = 0.0;
    }
    file1.close();
    return dataPoints;
}


void exportCsv(const std::string& fileName, float * dataPoints){
    std::ofstream file(fileName);
    if (!file.is_open()) {
        std::cerr << "Error opening file." << std::endl;
        return;
    }
    for (int i = 0;i<N;i++) {
        file << dataPoints[4*i] << "," << dataPoints[4*i+1] << "," << dataPoints[4*i+2] << "," << dataPoints[4*i+3] << "\n";
    }
    file.close();

}

float *randomCentroids(float* dataPoints){
    std::random_device dev;
    std::mt19937 rng(dev());
    std::uniform_int_distribution<std::mt19937::result_type> dist6(0,N-1);

    float * centroids = (float*) malloc(sizeof(float)*K*4);
    for(int i = 0; i < K; ++i){
        int index = dist6(rng);
        float x = dataPoints[index*4];
        float y = dataPoints[index*4+1];
        float z = dataPoints[index*4+2];
        centroids[4*i] = x;
        centroids[4*i+1] = y;
        centroids[4*i+2] = z;
        centroids[4*i+3]  = 0.0;
        
    }
    return centroids;
}



__device__ float euclideanDistance(float x1, float y1,float z1, float x2, float y2,float z2){
	return sqrt((x2-x1)*(x2-x1) + (y2-y1)*(y2-y1) + (z2-z1)*(z2-z1));
}

__global__ void centroidAssign(float *dataPoints_dev, float *centroids_dev){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index >= A) return;
	float minDistance = INFINITY;
	int clusterLabel = 0;
	for(int j = 0; j < K; ++j){
		float distance = euclideanDistance(dataPoints_dev[index*4],dataPoints_dev[index*4+1],dataPoints_dev[index*4+2],centroids_dev[j*4],centroids_dev[j*4+1],centroids_dev[j*4+2]);        
        if(distance < minDistance){
			minDistance = distance;
			clusterLabel = j;
		}
	}
	dataPoints_dev[index*4+3]=clusterLabel;
}






__global__ void centroidUpdate1(float *dataPoints_dev,float *centroids_dev){
	

	const int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index >= 4*A) return;
	
	__shared__ float dataPoints_shared[THREAD_PER_BLOCK];
	dataPoints_shared[threadIdx.x]= dataPoints_dev[index];


	if(index < 4*K){centroids_dev[index] = 0.0;}


	__syncthreads();
	if(threadIdx.x==0){
		float xSum[K],ySum[K],zSum[K],clusterCardinality[K];

		for(int i = 0; i < blockDim.x; ++i){
            int clust_id = dataPoints_shared[4*i + 3];
            xSum[clust_id]+=dataPoints_shared[4*i];
			ySum[clust_id]+=dataPoints_shared[4*i+1];
            zSum[clust_id]+=dataPoints_shared[4*i+2];
			clusterCardinality[clust_id] += 1.0;
		}

		for(int i = 0;i < K; i++){
			atomicAdd(&centroids_dev[i*4],xSum[i]);
			atomicAdd(&centroids_dev[i*4+1],ySum[i]);
            atomicAdd(&centroids_dev[i*4+2],zSum[i]);
			atomicAdd(&centroids_dev[i*4+3],clusterCardinality[i]);
		}
	}
	__syncthreads();

	if(index < K){
		centroids_dev[index*4] = centroids_dev[index*4]/centroids_dev[index*4+3];
		centroids_dev[index*4+1] = centroids_dev[index*4+1]/centroids_dev[index*4+3];
        centroids_dev[index*4+2] = centroids_dev[index*4+2]/centroids_dev[index*4+3];  
		centroids_dev[index*4+3] = 0.0; 

	}


}

int main(int argc, char* argv[]){

    float *dataPoints = loadCsv("ciao.csv");
    N = 1000;
    float *centroids = randomCentroids(dataPoints);
	
	//float *dataPoints = (float*) malloc(sizeof(float)*3*N);
    //float *centroids = (float*)malloc(sizeof(float)*K*4);


	/*for(int i = 0;i <N ; i++){ std::cout << dataPoints[4*i] << " " << dataPoints[4*i+1] << " " << dataPoints[4*i+2] << " " << dataPoints[4*i+3] << std::endl;}
	std::cout << "ciao" << std::endl;
    */

    /*centroids[0] = 2.39014695;
	centroids[1] = -0.57684421;
	centroids[2] = -10;
    centroids[3] = 0.0;
    centroids[4] = -7.53619806;
	centroids[5] = 4.49955772;
	centroids[6] = 13.4;
    centroids[7] = 0.0;*/



    /*std::cout << "data points: " << std::endl;
    for(int i = 0; i < N; ++i){ std::cout << dataPoints[i*3] << std::endl;}
    std::cout << "centroids: " << std::endl;
    for(int i = 0; i < K; ++i){ std::cout << centroids[i*3] << std::endl;}
    std::cout << "end printing" << std::endl;*/
 

    float *dataPoints_dev;
	hipMalloc(&dataPoints_dev, 4*N*sizeof(float));
    hipMemcpy(dataPoints_dev,dataPoints,4*N*sizeof(float),hipMemcpyHostToDevice);

    float *centroids_dev;
	hipMalloc(&centroids_dev,4*K*sizeof(int));
    hipMemcpy(centroids_dev,centroids,4*K*sizeof(float),hipMemcpyHostToDevice);


    std::cout << (N*4+THREAD_PER_BLOCK-1)/THREAD_PER_BLOCK << std::endl;

    for (int i = 0;i<100;i++){
        centroidAssign<<<(N*4+THREAD_PER_BLOCK-1)/THREAD_PER_BLOCK,THREAD_PER_BLOCK>>>(dataPoints_dev,centroids_dev);


        
        hipMemcpy(dataPoints,dataPoints_dev,4*N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(centroids,centroids_dev,4*K*sizeof(float),hipMemcpyDeviceToHost);
        /*std::cout << "cluster labels: " << std::endl;
        for(int i = 0; i < N; ++i){ std::cout << dataPoints[4*i+3] << ", ";}
        std::cout << "\nend printing" << std::endl;*/
        //cudaDeviceSynchronize();
        

        
        centroidUpdate1<<<(N*4+THREAD_PER_BLOCK-1)/THREAD_PER_BLOCK,THREAD_PER_BLOCK>>>(dataPoints_dev,centroids_dev);
        

        

    }
    hipMemcpy(dataPoints,dataPoints_dev,4*N*sizeof(float),hipMemcpyDeviceToHost);
    std::cout << "----------------------------cluster labels: -----------------" << std::endl;
    for(int i = 0; i < N; ++i){ std::cout << dataPoints[4*i+3] << ", ";}
    std::cout << "\n--------------------end printing-----------------------" << std::endl;

    exportCsv("fine.csv",dataPoints);


	return 0;
}
