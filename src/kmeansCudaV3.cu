#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <ctime>
#include <chrono>
#include "utils.h"

#ifndef K
#define K 5
#endif

#define THREAD_PER_BLOCK 1024

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__,#value, value)
static void CheckCudaErrorAux (const char *file, unsigned line, const char*statement, hipError_t err) {
    if (err == hipSuccess) return;
    std::cerr << statement<<" returned " << hipGetErrorString(err) <<"("<<err<< ") at "<<file<<":"<<line << std::endl;
    exit (1);
}

__device__ float distanceMetric(float x1, float y1, float x2, float y2){
    return (x2-x1)*(x2-x1) + (y2-y1)*(y2-y1); // Squared euclidean distance
}

x

    for(int i = localIndex; i < K; i+= blockDim.x) {
        atomicAdd(&(newCentroids_dev[i*2]), newCentroids_shared[i*2]);
        atomicAdd(&(newCentroids_dev[i*2+1]), newCentroids_shared[i*2+1]);
        atomicAdd(&(clusterCardinality_dev[i]), clusterCardinality_shared[i]);
    }
}

int main(int argc, char const *argv[]) {
    if(argc < 3) return -1;
    int N = getLineNumber(argv[1]);
    float *dataPoints = loadCsv(argv[1]);
    float *centroids = loadCsv(argv[2]);

    auto wcts = std::chrono::system_clock::now(); // Wall clock

    float *dataPoints_dev;
    CUDA_CHECK_RETURN(hipMalloc(&dataPoints_dev, 2*N*sizeof(float)));
    CUDA_CHECK_RETURN(hipMemcpy(dataPoints_dev, dataPoints, N*2*sizeof(float), hipMemcpyHostToDevice));

    float *centroids_dev;
    CUDA_CHECK_RETURN(hipMalloc(&centroids_dev, 2*K*sizeof(float)));
    CUDA_CHECK_RETURN(hipMemcpy(centroids_dev, centroids, K*2*sizeof(float), hipMemcpyHostToDevice));

    int *clusterLabel = (int*) malloc(sizeof(int)*N);
    int *clusterLabel_dev;
    CUDA_CHECK_RETURN(hipMalloc(&clusterLabel_dev, N*sizeof(int)));

    float *newCentroids = (float*)malloc(sizeof(float)*K*2);
    float *newCentroids_dev;
    CUDA_CHECK_RETURN(hipMalloc(&newCentroids_dev, K*2*sizeof(float)));

    int * clusterCardinality = (int*) malloc(sizeof(int)*K);
    int *clusterCardinality_dev;
    CUDA_CHECK_RETURN(hipMalloc(&clusterCardinality_dev, K*sizeof(int)));

    const int gridSize = (N + THREAD_PER_BLOCK - 1)/THREAD_PER_BLOCK;
    const int blockSize = THREAD_PER_BLOCK;

    for(int iter = 0; iter < 100; ++iter){
        CUDA_CHECK_RETURN(hipMemset(newCentroids_dev, 0.0f, 2 * K * sizeof(float)));
        CUDA_CHECK_RETURN(hipMemset(clusterCardinality_dev, 0, K * sizeof(int)));

        centroidAssignAndUpdate<<<gridSize, blockSize>>>(dataPoints_dev,centroids_dev,newCentroids_dev,clusterCardinality_dev,clusterLabel_dev,N);

        CUDA_CHECK_RETURN(hipMemcpy(newCentroids, newCentroids_dev, K*2*sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK_RETURN(hipMemcpy(clusterCardinality, clusterCardinality_dev, K*sizeof(int), hipMemcpyDeviceToHost));
        for (int i = 0; i < K; ++i) {
            int cardinality = clusterCardinality[i];
            if (cardinality <= 0) continue; 
            centroids[i*2] = newCentroids[i*2] / cardinality;
            centroids[i*2+1] = newCentroids[i*2+1] / cardinality;
        }
        CUDA_CHECK_RETURN(hipMemcpy(centroids_dev, centroids, K*2*sizeof(float), hipMemcpyHostToDevice));
    }
    CUDA_CHECK_RETURN(hipMemcpy(clusterLabel, clusterLabel_dev, N*sizeof(int), hipMemcpyDeviceToHost));

    std::chrono::duration<float> wctduration = (std::chrono::system_clock::now() - wcts);
    std::cout << wctduration.count() << std::endl;
    
#ifdef PRINT_FINAL_CENTROIDS
    std::cout << "Final centroids: " << std::endl; 
    for(int i=0;i<K; ++i){ std::cout << centroids[i*2] << ", " << centroids[i*2+1] << std::endl; }
#endif    

#ifdef EXPORT_FINAL_RESULT
    exportCsv("path/cudares.csv",dataPoints, clusterLabel, N);
    std::cout << "Done exporting result to csv. " << std::endl; 
#endif

    free(dataPoints); free(centroids);
    free(newCentroids); free(clusterCardinality);
    hipFree(dataPoints_dev); hipFree(centroids_dev);
    hipFree(newCentroids_dev); hipFree(clusterCardinality_dev);

    return 0;
}
